#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "error_check.h"
#include "cuda_pdist.cuh"
#include <iostream>
using namespace std;

#define P_(i_,k_) (point[i_*DIM+k_])
#define ADDR_(r_,c_) ( static_cast<size_t>(2*MAX_N-3-(r_))*(r_)>>1)-1+(c_)
#define R_(r_,c_) (result[ADDR_(r_,c_)]) 

#define SIZE 16

void pdist_CUDA(int &DONE, const int MAX_N, const int DIM, const int GPU_N, 
	       hipStream_t *stream, float *result, float *point_gpu[], float *result_gpu[])
{
	
	int i=0;
	int START=0;
	int END=0;
	int TransferSize=0;
	int S,E;
// 	DONE=8;
	while(DONE < MAX_N)
	{
		checkCudaErrors( hipSetDevice(i) );//Set device
		hipStreamSynchronize(stream[i]);
		
		#pragma omp critical(TASK)
		{
			START=DONE;
			DONE+=CHUNK;
			END=DONE;
// 			cout << "proceeding " << START << " to " << END << " on GPU " << i << endl;
		}
		if(END > MAX_N) END=MAX_N;
		if(START > MAX_N) break;
		
		launch_pdist_gpu( result_gpu[i], point_gpu[i], DIM, MAX_N, START, END, stream[i] );
		getLastCudaError("pdist_gpu() execution failed.\n");


		S=ADDR_(START,START+1);
		E=ADDR_(END-1,MAX_N-1);
		TransferSize=E-S+1;

		checkCudaErrors( hipMemcpyAsync(result+ADDR_(START,START+1),result_gpu[i], sizeof(float)*TransferSize,hipMemcpyDeviceToHost,stream[i]) );
	}
}

__global__ void pdist_gpu(float *out, float* in, int m, const int MAX_N, const int START, const int END)
{
	__shared__ float Ys[SIZE][SIZE];
	__shared__ float Xs[SIZE][SIZE];
	
	int bx= blockIdx.x;
	int by= blockIdx.y;
	
	int tx= threadIdx.x;
	int ty= threadIdx.y;
	
	int yBegin= (START*m) + by * SIZE * m;
	int xBegin=bx * SIZE * m;
	
	int yEnd = yBegin + m -1, y, x, k, o;
	float tmp, s=0;
	
	float dim_bound=0;
	int xIndex=bx*SIZE;
	int yIndex=START+by*SIZE;
	
	int addr_y= START+by*SIZE + ty;
	int addr_x= bx*SIZE + tx;
	
	
	if(  START/SIZE+by <= bx)
	{
		for (y=yBegin, x=xBegin; y<=yEnd ; x+=SIZE, y+=SIZE, dim_bound+=SIZE)
		{
			Ys[tx][ty]=0;
			Xs[tx][ty]=0;
			__syncthreads();
			
			if(dim_bound+tx < m && (yIndex+ty < END) )
				Ys[ty][tx] = in[y+(ty*m)+tx];
						
			if(dim_bound+tx < m && (xIndex+ty < MAX_N) )
				Xs[tx][ty] = in[x+(ty*m)+tx];
			__syncthreads();
			
			if ( START+by*SIZE+ty < bx*SIZE+tx ) //half
			{
				for( k = 0 ; k < SIZE ; k++)
				{
					tmp=Ys[ty][k] - Xs[k][tx];
					s+=tmp*tmp;
				}
			}
			__syncthreads();
		}
		o= (ADDR_(addr_y, addr_x))-(ADDR_(START,START+1));//
		
		if (addr_y < addr_x && addr_x < MAX_N && addr_y < END)
			out[o] = sqrtf(s);
	}
}

void launch_pdist_gpu(float *output, float* input, const int DIM, const int MAX_N, const int START, const int END, hipStream_t &stream)
{
	int TASK=END-START;
	dim3 grid((MAX_N%SIZE == 0 ? 0 : 1)+MAX_N/SIZE, (TASK%SIZE == 0 ? 0 : 1)+TASK/SIZE);
	dim3 threads(SIZE,SIZE);
	pdist_gpu<<< grid, threads, 0, stream>>>( output, input, DIM, MAX_N, START, END);
}

